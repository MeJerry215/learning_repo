#include "hip/hip_runtime.h"
#include "common.h"

__global__ void HelloWorldFromGPU() {
    printf("hello world from GPU!\n");
}

int main(int argc, char** argv) {
    printf("hello world from CPU\n");
    // hipSetDevice(0);
    HelloWorldFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    // hipDeviceReset();
    // hipMemcpyHostToDevice
}