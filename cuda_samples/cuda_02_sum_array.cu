#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "common.h"
void SumArrayOnHost(float *a, float *b, float *c, const int n)
{
    for (int i = 0; i < n; i++)
    {
        c[i] = a[i] + b[i];
    }
}

void init_data(float *p, int size)
{
    srand(42);
    for (int i = 0; i < size; i++)
    {
        p[i] = (float)(rand() & 0xff) / 10.0f;
    }
}

bool check_result(float* c, float* e, const int n) {
    for(int i = 0; i < n; i++) {
        if (fabs(c[i] - e[i]) > 1e-3) return false;
    }
    return true;
}

__global__ void SumArrayGpu(float *a, float *b, float *c, const int n)
{
    int tx = threadIdx.x, bx = blockIdx.x;
    int tid = blockDim.x * bx + tx;
    if (tid >= n)  return;
    c[tid] = a[tid] + b[tid];
}

int main(int argc, char** argv) {
    const int N = 1024;
    size_t nBytes = N * sizeof(float);
    float *h_a, *h_b, *h_c, *e_c;
    float *d_a, *d_b, *d_c;
    h_a = (float *)malloc(nBytes);
    h_b = (float *)malloc(nBytes);
    h_c = (float *)malloc(nBytes);
    e_c = (float *)malloc(nBytes);
    init_data(h_a, N);
    init_data(h_b, N);
    hipMalloc((void**)&d_a, nBytes);
    hipMalloc((void**)&d_b, nBytes);
    hipMalloc((void**)&d_c, nBytes);
    hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);
    SumArrayOnHost(h_a, h_b, e_c, N);
    SumArrayGpu<<<32, 32>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, nBytes, hipMemcpyDeviceToHost);
    if (check_result(h_c, e_c, N)) {
        printf("result equal\n");
    } else {
        printf("result not same\n");
    }
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(e_c);
}